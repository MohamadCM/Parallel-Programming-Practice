#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>

#define N 10000000
#define THREADS_PER_BLOCK 256
#define FACTOR 8

hipError_t addWithCuda(int* c, int* a, int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    for (size_t j = 0; j < FACTOR; j++)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x + j * (N / FACTOR);
        if (i < N)
            c[i] = a[i] + b[i];
    }
}
int main()
{
    int *a = (int *) malloc(N * sizeof(int));
    int *b = (int *) malloc(N * sizeof(int));
    int *c = (int *)  malloc(N * sizeof(int));
    srand(0);
    for (size_t i = 0; i < N; i++) {
        a[i] = rand() % 5;
        b[i] = rand() % 5;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    //printf("%d + %d = %d", a[1] + b[1], c[1]);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, int* a, int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    unsigned __int64 start = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    addKernel <<<ceil(size / (FACTOR * THREADS_PER_BLOCK)), THREADS_PER_BLOCK >>> (dev_c, dev_a, dev_b);

    hipDeviceSynchronize();
    unsigned __int64 end = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    printf("Elapsed time in milliseconds = %d\n", end - start);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
