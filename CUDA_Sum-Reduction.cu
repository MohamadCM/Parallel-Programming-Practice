#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

#define N 50000
#define block_size 32

hipError_t reduceWithCuda(int *input, int *output, unsigned int size);

__global__ void reduce0(int* g_idata, int* g_odata) {
    extern __shared__ int sdata[block_size];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main()
{
    int a[N];
    for (size_t i = 0; i < N; i++) {
        a[i] = i + 1;
    }
    int c[N] = { 0 };

    // Reduce vector in parallel.
    hipError_t cudaStatus = reduceWithCuda(a, c, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "reduce0Kernel failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t reduceWithCuda(int *input, int *output, unsigned int size)
{
    unsigned __int64 startTime = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    int *dev_i = 0;
    int *dev_o = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_o, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_i, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_i, input, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    const int num_blocks = (size / block_size) + ((size % block_size) ? 1 : 0);
    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    // Launch a kernel on the GPU with one thread for each element.
    reduce0<<<num_blocks, block_size >>>(dev_i, dev_o);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "reduce0Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduce0Kernel!\n", cudaStatus);
        goto Error;
    }
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    // float msecTotal = 0.0f;
    // hipEventElapsedTime(&msecTotal, start, stop);
    

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_o, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    int sum = 0;
    for (size_t i = 0; i < size; i++) {
        sum += output[i];
    }
    printf("Reduce calculation result = %d\n", sum);
    // printf("Kernel Elapsed time in milliseconds = %f\n", msecTotal);
    unsigned __int64 endTime = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    printf("Full elapsed time in milliseconds = %d\n", endTime - startTime);
Error:
    hipFree(dev_o);
    hipFree(dev_i);
    
    return cudaStatus;
}
